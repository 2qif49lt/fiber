#include "hip/hip_runtime.h"
#include <chrono>
#include <cstdlib>
#include <iostream>
#include <memory>
#include <random>
#include <tuple>

#include <hip/hip_runtime.h>

#include <boost/assert.hpp>
#include <boost/bind.hpp>
#include <boost/intrusive_ptr.hpp>

#include <boost/fiber/all.hpp>
#include <boost/fiber/cuda/waitfor.hpp>

__global__
void kernel( int size, int * a, int * b, int * c) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( idx < size) {
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;
        float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
        c[idx] = (as + bs) / 2;
    }
}

int main() {
    try {
        bool done = false;
        boost::fibers::fiber f1([&done]{
            std::cout << "f1: entered" << std::endl;
            try {
                hipStream_t stream;
                hipStreamCreate( & stream);
                int size = 1024 * 1024;
                int full_size = 20 * size;
                int * host_a, * host_b, * host_c;
                hipHostAlloc( & host_a, full_size * sizeof( int), hipHostMallocDefault);
                hipHostAlloc( & host_b, full_size * sizeof( int), hipHostMallocDefault);
                hipHostAlloc( & host_c, full_size * sizeof( int), hipHostMallocDefault);
                int * dev_a, * dev_b, * dev_c;
                hipMalloc( & dev_a, size * sizeof( int) );
                hipMalloc( & dev_b, size * sizeof( int) );
                hipMalloc( & dev_c, size * sizeof( int) );
                std::minstd_rand generator;
                std::uniform_int_distribution<> distribution(1, 6);
                for ( int i = 0; i < full_size; ++i) {
                    host_a[i] = distribution( generator);
                    host_b[i] = distribution( generator);
                }
                for ( int i = 0; i < full_size; i += size) {
                    hipMemcpyAsync( dev_a, host_a + i, size * sizeof( int), hipMemcpyHostToDevice, stream);
                    hipMemcpyAsync( dev_b, host_b + i, size * sizeof( int), hipMemcpyHostToDevice, stream);
                    kernel<<< size / 256, 256, 0, stream >>>( size, dev_a, dev_b, dev_c);
                    hipMemcpyAsync( host_c + i, dev_c, size * sizeof( int), hipMemcpyDeviceToHost, stream);
                }
                auto result = boost::fibers::cuda::waitfor_all( stream);
                BOOST_ASSERT( stream == std::get< 0 >( result) );
                BOOST_ASSERT( hipSuccess == std::get< 1 >( result) );
                hipHostFree( host_a);
                hipHostFree( host_b);
                hipHostFree( host_c);
                hipFree( dev_a);
                hipFree( dev_b);
                hipFree( dev_c);
                hipStreamDestroy( stream);
                done = true;
            } catch ( std::exception const& ex) {
                std::cerr << "exception: " << ex.what() << std::endl;
            }
            std::cout << "f1: leaving" << std::endl;
        });
        boost::fibers::fiber f2([&done]{
            std::cout << "f2: entered" << std::endl;
            while ( ! done) {
                std::cout << "f2: sleeping" << std::endl;
                boost::this_fiber::sleep_for( std::chrono::milliseconds( 1 ) );
            }
            std::cout << "f2: leaving" << std::endl;
        });
        f1.join();
        f2.join();
        std::cout << "done." << std::endl;
        return EXIT_SUCCESS;
    } catch ( std::exception const& e) {
        std::cerr << "exception: " << e.what() << std::endl;
    } catch (...) {
        std::cerr << "unhandled exception" << std::endl;
    }
	return EXIT_FAILURE;
}
